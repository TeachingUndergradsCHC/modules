/*
 * CUDA program to multiply matrices (fills in matrices itself).
 * This version uses tiling to improve the memory performance.
 * 
 * IT IS INCOMPLETE; THE TODO PART BELOW NEEDS TO BE FILLED IN
 * 
 * compile with:
 *      nvcc -o tiled_matrix_multiply tiled_matrix_multiply.cu
 *
 * run with:
 *      ./tiled_matrix_multiply
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cassert>
#include <cstdlib>

//constants to control the program:
#define NTESTS 1           /* # of tests to run */
#define TILE_WIDTH 32      /* # of threads in each dimension per block */
                           /* #threads per block = TILE_WIDTH * TILE_WIDTH */
#define WIDTH 1024         /* matrix dimensions (assumes square matrix) */

__global__ void tiledkernel(float* Md, float* Nd, float* Pd, int width) {
  //method to run on GPU; called once per element of output matrix

  //allocate shared memory (shared between all threads of a block) to hold 1 tile of each matrix
  __shared__ float Mds[TILE_WIDTH][TILE_WIDTH];  //holds tile sharing row with element
  __shared__ float Nds[TILE_WIDTH][TILE_WIDTH];  //holds tile sharing col with element

  //set up short names for indices
  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  //calculate indices for the element to compute:
  int row = by * TILE_WIDTH + ty;
  int col = bx * TILE_WIDTH + tx;

  float tmp = 0;  //local variable in which to accumulate the answer

  int num_tiles = (width+TILE_WIDTH-1)/TILE_WIDTH;  //width of matrix in tiles (rounded up)
  for (int m=0; m < num_tiles; m++) {  //loop over tiles in row and column containing element

    //load Mds and Nds; this thread loads value at its postion; other threads in block load other values
    //load 0s for values outside the matrix (tiles can be partially in and partially out of matrix)
    if (m*TILE_WIDTH + tx < width && row < width)
      Mds[ty][tx] = Md[row*width + (m*TILE_WIDTH + tx)];
    else
      Mds[ty][tx] = 0.0;
    if (m*TILE_WIDTH + ty < width && col < width)
      Nds[ty][tx] = Nd[(m*TILE_WIDTH + ty) * width + col];
    else
      Nds[ty][tx] = 0.0;

    __syncthreads();  //barrier to wait for other threads before using Mds and Nds

    //TODO: Add the contribution of Mds and Nds to tmp

    __syncthreads();  //another barrier; wait for all threads to use Mds and Mds before replacing them
  }

  //put answer into the result matrix
  if (row < width && col < width)
    Pd[row*width+col] = tmp;
}

void verify_solution(float *a, float *b, float *c, int N) {
  //verify the solution on the CPU

  //threshold for matching: (0 ok since all vals are small ints)
  float epsilon = 0;  

  for (int i = 0; i < N; i++) {      //for every column...
    for (int j = 0; j < N; j++) {    //for every row in that column
      float tmp = 0;
      for (int k = 0; k < N; k++) {
        tmp += a[i * N + k] * b[k * N + j];
      }

    // Check against the GPU result, throw an error if not equal 
    assert(fabs(c[i * N + j] - tmp) <= epsilon);
    }
  }
}

void check(hipError_t retVal) {
  //takes return value of a CUDA function and checks if it was an error

  if(retVal != hipSuccess) {
    if (retVal==hipErrorInvalidConfiguration)
      printf("Number of Threads per block is not valid");
    fprintf(stderr, "ERROR: %s\n", hipGetErrorString(retVal));
    exit(1);
  }
}

float runTest(float* M, float* N, float* P, float* Md, float* Nd, float* Pd, int size) {

  //allocate timers
  hipEvent_t start;
  check(hipEventCreate(&start));
  hipEvent_t stop;
  check(hipEventCreate(&stop));

  //start timer
  check(hipEventRecord(start,0));

  //copy data from host to device
  check(hipMemcpy(Md, M, size, hipMemcpyHostToDevice));
  check(hipMemcpy(Nd, N, size, hipMemcpyHostToDevice));

  //call the kernel
  int gridsize = (WIDTH+TILE_WIDTH-1)/TILE_WIDTH;
  dim3 dimGrid(gridsize, gridsize);
  dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
  tiledkernel<<<dimGrid,dimBlock>>>(Md, Nd, Pd, WIDTH);

  //check if kernel encountered an error due to invalid configurations
  hipError_t err = hipGetLastError();
  check(err);

  //transfer result matrix to the host
  check(hipMemcpy(P, Pd, size, hipMemcpyDeviceToHost));

  //stop timer and store time
  check(hipEventRecord(stop,0));
  check(hipEventSynchronize(stop));
  float diff;
  check(hipEventElapsedTime(&diff, start, stop));

  //deallocate timers
  check(hipEventDestroy(start));
  check(hipEventDestroy(stop));
  
  //print and return time
  printf("Time : %f ms\n", diff);
  return diff;
}

int main() {
  float* M;       //input arrays (on host)
  float* N;
  float* P;       //output array (on host)

  float* Md;      //input arrays (on device)
  float* Nd;
  float* Pd;      //output array (on device)
  
  int size = WIDTH * WIDTH * sizeof(float);  //size of matrix in bytes
  
  //allocate memory
  M = (float*) malloc(size);
  N = (float*) malloc(size);
  P = (float*) malloc(size);
  check(hipMalloc((void**) &Md, size));
  check(hipMalloc((void**) &Nd, size));
  check(hipMalloc((void**) &Pd, size));

  //fill M and N arrays (all elements <= 2048 so results stay small)
  int cor = 0;
  for(int i=0; i < WIDTH * WIDTH; i++){
    M[i] = N[i] = i-cor ;
    if(i % 2048 == 0)
        cor=i;
  }
  
  float total_time = 0;  //accumultate execution times for averaging

  for(int i=0; i < NTESTS; i++)
    total_time += runTest(M, N, P, Md, Nd, Pd, size);
  
  printf("Avg for %d tests: %f ms and size of matrix %d\n",
	 NTESTS, total_time/(float)NTESTS, WIDTH);
  
  verify_solution(M,N,P,WIDTH);  //verify result 

  //free all memory:
  free(M);
  free(N);
  free(P);
  check(hipFree(Md));
  check(hipFree(Nd));
  check(hipFree(Pd));
}
