/*
 * Sample program that uses CUDA to perform element-wise add of two
 * vectors.  Each element is the responsibility of a separate thread.
 *
 * compile with:
 *    nvcc -o addVectors addVectors.cu
 * run with:
 *    ./addVectors
 */


#include <hip/hip_runtime.h>
#include <stdio.h>

//problem size (vector length):
#define N 10

__global__ void kernel(int* res, int* a, int* b) {
  //function that runs on GPU to do the addition
  //sets res[i] = a[i] + b[i]; each thread is responsible for one value of i

  int thread_id = threadIdx.x + blockIdx.x*blockDim.x;
  if(thread_id < N) {
    res[thread_id] = a[thread_id] + b[thread_id];
  }
}

void check(hipError_t retVal) {
  //takes return value of a CUDA function and checks if it was an error
  if(retVal != hipSuccess) {
    fprintf(stderr, "ERROR: %s\n", hipGetErrorString(retVal));
    exit(1);
  }
}

int main() {
  int* a;       //input arrays (on host)
  int* b;
  int* res;     //output array (on host)

  int* a_dev;   //input arrays (on GPU)
  int* b_dev;
  int* res_dev; //output array (on GPU) 

  //allocate memory
  a = (int*) malloc(N*sizeof(int));
  b = (int*) malloc(N*sizeof(int));
  res = (int*) malloc(N*sizeof(int));
  check(hipMalloc((void**) &a_dev, N*sizeof(int)));
  check(hipMalloc((void**) &b_dev, N*sizeof(int)));
  check(hipMalloc((void**) &res_dev, N*sizeof(int)));

  //set up contents of a and b
  for(int i=0; i < N; i++)
    a[i] = b[i] = i;

  //allocate timers
  hipEvent_t start;
  check(hipEventCreate(&start));
  hipEvent_t stop;
  check(hipEventCreate(&stop));

  //start timer
  check(hipEventRecord(start,0));

  //transfer a and b to the GPU
  check(hipMemcpy(a_dev, a, N*sizeof(int), hipMemcpyHostToDevice));
  check(hipMemcpy(b_dev, b, N*sizeof(int), hipMemcpyHostToDevice));

  //call the kernel
  int threads = 512;                   //# threads per block
  int blocks = (N+threads-1)/threads;  //# blocks (N/threads rounded up)
  kernel<<<blocks,threads>>>(res_dev, a_dev, b_dev);

  //transfer res to the host
  check(hipMemcpy(res, res_dev, N*sizeof(int), hipMemcpyDeviceToHost));

  //stop timer and print time
  check(hipEventRecord(stop,0));
  check(hipEventSynchronize(stop));
  float diff;
  check(hipEventElapsedTime(&diff, start, stop));
  printf("time: %f ms\n", diff);

  //deallocate timers
  check(hipEventDestroy(start));
  check(hipEventDestroy(stop));

  //verify results
  for(int i=0; i < N; i++)
    printf("%d ", res[i]);
  printf("\n");

  //free the memory
  free(a);
  free(b);
  free(res);
  check(hipFree(a_dev));
  check(hipFree(b_dev));
  check(hipFree(res_dev));
}
